// Name: U.H. Anuji de Silva
// Student Id: 1432292


#include <hip/hip_runtime.h>
#include <stdio.h>
//inserting stdlib.h library to run Cuda Malloc
#include <stdlib.h>
#define N 4
 
 
int main()
{
   int i, j =0;

  int A[N][N] =
    {
      {1, 5, 6, 7},
      {4, 4, 8, 0},
      {2, 3, 4, 5},
      {2, 3, 4, 5}
   };

  int B[N][N] = 
    {
      {1, 5, 6, 7},
      {4, 4, 8, 0},
      {2, 3, 4, 5},
      {2, 3, 4, 5}
   };

  int C[N][N] = 
     {
      {0, 0, 0, 0},
      {0, 0, 0, 0},
      {0, 0, 0, 0},
      {0, 0, 0, 0}
   };

 
     for(i=0;i<N;i++){
        for(j=0;j<N;j++){
            C[i][j] = A[i][j] + B[i][j];
        }
    }
 
   printf("Sum of entered matrices:-\n");
 
    for(i=0;i<N;i++){
        for(j=0;j<N;j++){
            printf("%d ", C[i][j]);
        }
        printf("\n");
    }
 
   return 0;
}
